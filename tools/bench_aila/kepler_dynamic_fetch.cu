#include "hip/hip_runtime.h"
/*
 *  Copyright (c) 2009-2011, NVIDIA Corporation
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are met:
 *      * Redistributions of source code must retain the above copyright
 *        notice, this list of conditions and the following disclaimer.
 *      * Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimer in the
 *        documentation and/or other materials provided with the distribution.
 *      * Neither the name of NVIDIA Corporation nor the
 *        names of its contributors may be used to endorse or promote products
 *        derived from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 *  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *  DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 *  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 *  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 *  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    GK104-optimized variant of the "Persistent speculative
    while-while" kernel used in:

    "Understanding the Efficiency of Ray Traversal on GPUs",
    Timo Aila and Samuli Laine,
    Proc. High-Performance Graphics 2009

    This variant fetches new work dynamically as soon as the
    warp occupancy drops below a pre-determined threshold.
*/

#include "CudaTracerKernels.hpp"

//------------------------------------------------------------------------

#define STACK_SIZE              64          // Size of the traversal stack in local memory.
#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

static __device__ int g_warpCounter;    // Work counter for persistent threads.

// Old Config ------------------------------------------------------------------------
/*
extern "C" __global__ void queryConfig(void)
{
    g_config.bvhLayout = BVHLayout_Compact2;
    g_config.blockWidth = 32;
    g_config.blockHeight = 4;
    g_config.usePersistentThreads = 1;
}*/

// New Config ------------------------------------------------------------------------

static constexpr int g_blockWidth  = 32;
static constexpr int g_blockHeight = 4;

__forceinline__ __device__ float prodsign(float x, float y) {
    return __int_as_float(__float_as_int(x) ^ (__float_as_int(y) & 0x80000000));
}

//------------------------------------------------------------------------

TRACE_FUNC
{
    // Traversal stack in CUDA thread-local memory.

    int traversalStack[STACK_SIZE];
    traversalStack[0] = EntrypointSentinel; // Bottom-most entry.

    // Live state during traversal, stored in registers.

    float   origx, origy, origz;            // Ray origin.
    char*   stackPtr;                       // Current position in traversal stack.
    int     leafAddr;                       // First postponed leaf, non-negative if none.
    //int     leafAddr2;                      // Second postponed leaf, non-negative if none.
    int     nodeAddr = EntrypointSentinel;  // Non-negative: current internal node, negative: second postponed leaf.
    int     hitIndex;                       // Triangle index of the closest intersection, -1 if none.
    float   hitT;                           // t-value of the closest intersection.
    float   hitU;
    float   hitV;
    float   tmin;
    int     rayidx;
    float   oodx;
    float   oody;
    float   oodz;
    float   dirx;
    float   diry;
    float   dirz;
    float   idirx;
    float   idiry;
    float   idirz;

    static constexpr unsigned all_mask = unsigned(-1);

    // Initialize persistent threads.

    __shared__ volatile int nextRayArray[MaxBlockHeight]; // Current ray index in global buffer.

    // Persistent threads: fetch and process rays in a loop.

    do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        // Fetch new rays from the global pool using lane 0.

        const bool          terminated     = nodeAddr==EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot_sync(all_mask, terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));

        if(terminated)
        {
            if (idxTerminated == 0)
                rayBase = atomicAdd(&g_warpCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
            if (rayidx >= numRays)
                break;

            // Fetch ray.

            float4 o = FETCH_GLOBAL(rays, rayidx * 2 + 0, float4);
            float4 d = FETCH_GLOBAL(rays, rayidx * 2 + 1, float4);
            origx = o.x;
            origy = o.y;
            origz = o.z;
            tmin  = o.w;
            dirx  = d.x;
            diry  = d.y;
            dirz  = d.z;
            hitT  = d.w;
            hitU = hitV = 0.0f;
            float ooeps = exp2f(-80.0f); // Avoid div by zero.
            idirx = 1.0f / (fabsf(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
            idiry = 1.0f / (fabsf(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
            idirz = 1.0f / (fabsf(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
            oodx  = origx * idirx;
            oody  = origy * idiry;
            oodz  = origz * idirz;

            // Setup traversal.

            stackPtr = (char*)&traversalStack[0];
            leafAddr = 0;   // No postponed leaf.
            //leafAddr2= 0;   // No postponed leaf.
            nodeAddr = 0;   // Start from the root.
            hitIndex = -1;  // No triangle intersected so far.
        }

        // Traversal loop.

        while(nodeAddr != EntrypointSentinel)
        {
            // Traverse internal nodes until all SIMD lanes have found a leaf.

//          while (nodeAddr >= 0 && nodeAddr != EntrypointSentinel)
            while (uint(nodeAddr) < uint(EntrypointSentinel))   // functionally equivalent, but faster
            {
                // Fetch AABBs of the two child nodes.
                const float4 n0xy = tex1Dfetch(t_nodesA, nodeAddr + 0); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
                const float4 n1xy = tex1Dfetch(t_nodesA, nodeAddr + 1); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
                const float4 nz   = tex1Dfetch(t_nodesA, nodeAddr + 2); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
                      float4 tmp  = tex1Dfetch(t_nodesA, nodeAddr + 3); // child_index0, child_index1
                      int2  cnodes= *(int2*)&tmp;

                // Intersect the ray against the child nodes.

                const float c0lox = n0xy.x * idirx - oodx;
                const float c0hix = n0xy.y * idirx - oodx;
                const float c0loy = n0xy.z * idiry - oody;
                const float c0hiy = n0xy.w * idiry - oody;
                const float c0loz = nz.x   * idirz - oodz;
                const float c0hiz = nz.y   * idirz - oodz;
                const float c1loz = nz.z   * idirz - oodz;
                const float c1hiz = nz.w   * idirz - oodz;
                const float c0min = spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin);
                const float c0max = spanEndKepler  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT);
                const float c1lox = n1xy.x * idirx - oodx;
                const float c1hix = n1xy.y * idirx - oodx;
                const float c1loy = n1xy.z * idiry - oody;
                const float c1hiy = n1xy.w * idiry - oody;
                const float c1min = spanBeginKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, tmin);
                const float c1max = spanEndKepler  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, hitT);

                bool swp = (c1min < c0min);

                bool traverseChild0 = (c0max >= c0min);
                bool traverseChild1 = (c1max >= c1min);

                // Neither child was intersected => pop stack.

                if (!traverseChild0 && !traverseChild1)
                {
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // Otherwise => fetch child pointers.

                else
                {
                    nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;

                    // Both children were intersected => push the farther one.

                    if (traverseChild0 && traverseChild1)
                    {
                        if (swp)
                            swap(nodeAddr, cnodes.y);
                        stackPtr += 4;
                        *(int*)stackPtr = cnodes.y;
                    }
                }

                // First leaf => postpone and continue traversal.

                if (nodeAddr < 0 && leafAddr  >= 0)     // Postpone max 1
//              if (nodeAddr < 0 && leafAddr2 >= 0)     // Postpone max 2
                {
                    //leafAddr2= leafAddr;          // postpone 2
                    leafAddr = nodeAddr;
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // All SIMD lanes have found a leaf? => process them.

                // NOTE: inline PTX implementation of "if(!__any(leafAddr >= 0)) break;".
                // tried everything with CUDA 4.2 but always got several redundant instructions.

                unsigned int mask;
                asm("{\n"
                    "   .reg .pred p;               \n"
                    "setp.ge.s32        p, %1, 0;   \n"
                    "vote.sync.ballot.b32    %0,p,0xffffffff;       \n"
                    "}"
                    : "=r"(mask)
                    : "r"(leafAddr));
                if(!mask)
                    break;

                //if(!__any(leafAddr >= 0))
                //    break;
            }

            // Process postponed leaf nodes.

            while (leafAddr < 0)
            {
                for (int triAddr = ~leafAddr;; triAddr += 3)
                {
                    // Original Woop Intersection ---------------------------------------------------------------
                    /*
                    // Tris in TEX (good to fetch as a single batch)
                    const float4 v00 = tex1Dfetch(t_trisA, triAddr + 0);
                    const float4 v11 = tex1Dfetch(t_trisA, triAddr + 1);
                    const float4 v22 = tex1Dfetch(t_trisA, triAddr + 2);

                    // End marker (negative zero) => all triangles processed.
                    if (__float_as_int(v00.x) == 0x80000000)
                        break;

                    float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;
                    float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);
                    float t = Oz * invDz;

                    if (t > tmin && t < hitT)
                    {
                        // Compute and check barycentric u.

                        float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;
                        float Dx = dirx*v11.x + diry*v11.y + dirz*v11.z;
                        float u = Ox + t*Dx;

                        if (u >= 0.0f)
                        {
                            // Compute and check barycentric v.

                            float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
                            float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
                            float v = Oy + t*Dy;

                            if (v >= 0.0f && u + v <= 1.0f)
                            {
                                // Record intersection.
                                // Closest intersection not required => terminate.

                                hitT = t;
                                hitIndex = triAddr;
                                if (anyHit)
                                {
                                    nodeAddr = EntrypointSentinel;
                                    break;
                                }
                            }
                        }
                    }*/
                    // Moeller Trumbore Intersection ----------------------------------------------------------------
                    const float4 v0 = tex1Dfetch(t_trisA, triAddr + 0);
                    const float4 e1 = tex1Dfetch(t_trisA, triAddr + 1);
                    const float4 e2 = tex1Dfetch(t_trisA, triAddr + 2);

                    float nx = e1.y * e2.z - e1.z * e2.y;
                    float ny = e1.z * e2.x - e1.x * e2.z;
                    float nz = e1.x * e2.y - e1.y * e2.x;
                    float cx = v0.x - origx;
                    float cy = v0.y - origy;
                    float cz = v0.z - origz;
                    float rx = diry * cz - dirz * cy;
                    float ry = dirz * cx - dirx * cz;
                    float rz = dirx * cy - diry * cx;
                    float det = nx * dirx + ny * diry + nz * dirz;
                    float abs_det = det < 0 ? -det : det;

                    float u = prodsign(rx * e2.x + ry * e2.y + rz * e2.z, det);
                    float v = prodsign(rx * e1.x + ry * e1.y + rz * e1.z, det);

                    if (u >= 0.0f && v >= 0.0f && abs_det >= u + v && abs_det != 0.0f) {
                        float t = prodsign(cx * nx + cy * ny + cz * nz, det);
                        if (t >= abs_det * tmin && t <= abs_det * hitT) {
                            float inv_det = 1.0f / abs_det;
                            hitT = t * inv_det;
                            hitU = u * inv_det;
                            hitV = v * inv_det;
                            hitIndex = triAddr;
                            if (anyHit) {
                                nodeAddr = EntrypointSentinel;
                                break;
                            }
                        }
                    }

                    if (__float_as_int(e2.w) & 0x80000000)
                        break;
                } // triangle

                // Another leaf was postponed => process it as well.

//              if(leafAddr2<0) { leafAddr = leafAddr2; leafAddr2=0; } else     // postpone2
                {
                    leafAddr = nodeAddr;
                    if (nodeAddr < 0)
                    {
                        nodeAddr = *(int*)stackPtr;
                        stackPtr -= 4;
                    }
                }
            } // leaf

            // DYNAMIC FETCH

            if( __popc(__ballot_sync(all_mask, true)) < DYNAMIC_FETCH_THRESHOLD )
                break;

        } // traversal

        // Remap intersected triangle index, and store the result.

        if (hitIndex == -1) { STORE_RESULT(rayidx, -1, hitT, 0, 0); }
        else                { STORE_RESULT(rayidx, __int_as_float(FETCH_TEXTURE(triIndices, hitIndex, int)), hitT, hitU, hitV); }
    } while(true);
}

// Kernel Launch ------------------------------------------------------------------------

#include <fstream>
#include <iostream>
#include <vector>
#include <cassert>

#include "traversal.h"

#define CHECK_CUDA_CALL(x) check_cuda_call(x, __FILE__, __LINE__)

__host__ static void check_cuda_call(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << file << "(" << line << "): " << hipGetErrorString(err) << std::endl;
        abort();
    }
}

static float4* cuda_nodes = nullptr;
static float4* cuda_tris  = nullptr;
static int*    cuda_ids   = nullptr;

void setup_traversal(const Node2* nodes, size_t num_nodes, const Tri1* tris, size_t num_tris) {
    assert(sizeof(Node2) == sizeof(float4) * 4);
    assert(sizeof(Tri1)  == sizeof(float4) * 3);
    assert(!cuda_nodes && !cuda_tris && !cuda_ids);

    CHECK_CUDA_CALL(hipMalloc(&cuda_nodes, sizeof(float4) * 4 * num_nodes));
    CHECK_CUDA_CALL(hipMalloc(&cuda_tris,  sizeof(float4) * 3 * num_tris));
    CHECK_CUDA_CALL(hipMalloc(&cuda_ids,   sizeof(int)    * 3 * num_tris));

    std::vector<int> ids(num_tris * 3);
    for (int i = 0; i < num_tris; i++) {
        ids[3 * i + 0] = tris[i].prim_id;
        ids[3 * i + 1] = tris[i].prim_id;
        ids[3 * i + 2] = tris[i].prim_id;
    }
    CHECK_CUDA_CALL(hipMemcpy(cuda_ids, ids.data(), sizeof(int) * 3 * num_tris, hipMemcpyHostToDevice));

    std::vector<float4> nodes_aila(num_nodes * 4);
    for (int i = 0; i < num_nodes; i++) {
        const auto& node = nodes[i];
        nodes_aila[i * 4 + 0] = make_float4(node.bounds[0], node.bounds[1], node.bounds[ 2], node.bounds[ 3]);
        nodes_aila[i * 4 + 1] = make_float4(node.bounds[4], node.bounds[5], node.bounds[ 6], node.bounds[ 7]);
        nodes_aila[i * 4 + 2] = make_float4(node.bounds[8], node.bounds[9], node.bounds[10], node.bounds[11]);
        // indexing is done on float4, not on nodes/tris
        union { int i; float f; } left { .i = node.child[0] < 0 ? ~((~node.child[0]) * 3) : (node.child[0] - 1) * 4 };
        union { int i; float f; } right{ .i = node.child[1] < 0 ? ~((~node.child[1]) * 3) : (node.child[1] - 1) * 4 };
        nodes_aila[i * 4 + 3] = make_float4(left.f, right.f, 0, 0);
    }
    CHECK_CUDA_CALL(hipMemcpy(cuda_nodes, nodes_aila.data(), sizeof(float4) * 4 * num_nodes, hipMemcpyHostToDevice));
    
    CHECK_CUDA_CALL(hipMemcpy(cuda_tris, tris, sizeof(float4) * 3 * num_tris, hipMemcpyHostToDevice));

    CHECK_CUDA_CALL(hipBindTexture(nullptr, t_nodesA,     cuda_nodes, sizeof(float4) * 4 * num_nodes));
    CHECK_CUDA_CALL(hipBindTexture(nullptr, t_trisA,      cuda_tris,  sizeof(float4) * 3 * num_tris ));
    CHECK_CUDA_CALL(hipBindTexture(nullptr, t_triIndices, cuda_ids,   sizeof(int)    * 3 * num_tris ));
}

void shutdown_traversal() {
    CHECK_CUDA_CALL(hipUnbindTexture(t_nodesA));
    CHECK_CUDA_CALL(hipUnbindTexture(t_trisA));
    CHECK_CUDA_CALL(hipUnbindTexture(t_triIndices));
    CHECK_CUDA_CALL(hipFree(cuda_tris));
    CHECK_CUDA_CALL(hipFree(cuda_ids));
    CHECK_CUDA_CALL(hipFree(cuda_nodes));
}

void bench_traversal(const Ray1* rays, Hit1* hits, int num_rays, double* timings, int ntimes, bool any) {
    assert(sizeof(Ray1) == sizeof(float4) * 2);
    assert(sizeof(Hit1) == sizeof(int4));

    float4* cuda_rays;
    int4*   cuda_hits;
    CHECK_CUDA_CALL(hipMalloc(&cuda_rays, sizeof(float4) * 2 * num_rays));
    CHECK_CUDA_CALL(hipMalloc(&cuda_hits, sizeof(int4) * num_rays));
    CHECK_CUDA_CALL(hipMemcpy(cuda_rays, rays, sizeof(float4) * 2 * num_rays, hipMemcpyHostToDevice));

    CHECK_CUDA_CALL(hipDeviceSynchronize());

    int num_blocks = 0;
    CHECK_CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, trace, g_blockWidth * g_blockHeight, MaxBlockHeight * sizeof(int)));
    int dev = 0;
    CHECK_CUDA_CALL(hipGetDevice(&dev));
    hipDeviceProp_t props;
    CHECK_CUDA_CALL(hipGetDeviceProperties(&props, dev));

    num_blocks *= props.multiProcessorCount;

    hipEvent_t start, end;
    CHECK_CUDA_CALL(hipEventCreate(&start));
    CHECK_CUDA_CALL(hipEventCreate(&end));

    dim3 grid(num_blocks, 1, 1);
    dim3 block(g_blockWidth, g_blockHeight, 1);
    for (int i = 0; i < ntimes; i++) {
        // Here, we assume that the memcpy can be hidden in a real world application
        // Therefore, it is not included in the performance measurement
        int zero = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter), &zero, sizeof(int));

        CHECK_CUDA_CALL(hipEventRecord(start));
        trace<<<grid, block>>>(
            num_rays,
            any,
            cuda_rays,
            cuda_hits,
            nullptr,
            nullptr,
            nullptr,
            nullptr,
            nullptr,
            nullptr,
            nullptr,
            nullptr
        );
        CHECK_CUDA_CALL(hipEventRecord(end));
        CHECK_CUDA_CALL(hipEventSynchronize(end));

        float ms;
        CHECK_CUDA_CALL(hipEventElapsedTime(&ms, start, end));
        if (timings) timings[i] = ms;
    }

    CHECK_CUDA_CALL(hipEventDestroy(start));
    CHECK_CUDA_CALL(hipEventDestroy(end));

    CHECK_CUDA_CALL(hipMemcpy(hits, cuda_hits, sizeof(int4) * num_rays, hipMemcpyDeviceToHost));

    CHECK_CUDA_CALL(hipFree(cuda_rays));
    CHECK_CUDA_CALL(hipFree(cuda_hits));
} 
