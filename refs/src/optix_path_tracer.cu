#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include <cfloat>

#include "optix_path_tracer.h"

#define PI 3.14159265359f

static constexpr float offset = 1e-4f; // Global ray distance offset to avoid self-intersections

struct RayState {
    uint rnd;
    uint depth;
    float3 contrib;
    float3 color;
    float mis;
    float3 next_org;
    float3 next_dir;
    bool done;
};

struct ShadowRayState {
    bool in_shadow;
};

struct MaterialSample {
    float pdf;
    float3 color;
    float cos;
    float3 dir;
};

// Variables -----------------------------------------------------------------------

// Attributes
rtDeclareVariable(float3,   attr_normal,      attribute normal     , );
rtDeclareVariable(float3,   attr_face_normal, attribute face_normal, );
rtDeclareVariable(float2,   attr_texcoord,    attribute texcoord   , );
rtDeclareVariable(int3,     attr_index,       attribute index      , );
rtDeclareVariable(uint,     attr_material,    attribute material   , );

// Per ray
rtDeclareVariable(optix::Ray, ray,          rtCurrentRay          , );
rtDeclareVariable(float,      ray_dist,     rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex         , );

// Globals
rtDeclareVariable(rtObject, top_object,        , );
rtDeclareVariable(float,    pdf_lightpick,     , );
rtDeclareVariable(uint,     frame_number,      , );
rtDeclareVariable(uint,     num_lights,        , );
rtDeclareVariable(uint,     max_path_depth,    , );
rtDeclareVariable(uint,     samples_per_pixel, , );
rtDeclareVariable(uint,     film_width,        , );
rtDeclareVariable(uint,     film_height,       , );

rtDeclareVariable(float3, cam_eye,   , );
rtDeclareVariable(float3, cam_dir,   , );
rtDeclareVariable(float3, cam_right, , );
rtDeclareVariable(float3, cam_up,    , );
rtDeclareVariable(float2, cam_dim,   , );

// Buffers
rtBuffer<float3>   frame_buffer;
rtBuffer<Material> materials;
rtBuffer<Light>    lights;

rtBuffer<float3> vertex_buffer;     
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int4>   index_buffer;

// Utility functions ---------------------------------------------------------------

__device__ int32_t xorshift(uint& seed) {
    auto x = seed;
    x = x == 0 ? 1 : x;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    seed = x;
    return x;
}

__device__ float randf(uint& rnd) {
    uint u = xorshift(rnd);
    return __int_as_float((127u << 23u) | (u & 0x7FFFFFu)) - 1.0f;
}

__device__ int32_t randi(uint& rnd) {
    return xorshift(rnd);
}

__device__ uint fnv_init() { return 0x811C9DC5u; }

__device__ uint fnv_hash(uint h, uint d) {
    h = (h * 16777619u) ^ ( d         & 0xFFu);
    h = (h * 16777619u) ^ ((d >>  8u) & 0xFFu);
    h = (h * 16777619u) ^ ((d >> 16u) & 0xFFu);
    h = (h * 16777619u) ^ ((d >> 24u) & 0xFFu);
    return h;
}

__device__ void gen_local_coords(const float3& normal, float3& tangent, float3& bitangent) {
    auto sign = normal.z >= 0.0f ? 1.0f : -1.0f;
    auto a = -1.0f / (sign + normal.z);
    auto b = normal.x * normal.y * a;

    tangent   = make_float3(1.0f + sign * normal.x * normal.x * a, sign * b, -sign * normal.x);
    bitangent = make_float3(b, sign + normal.y * normal.y * a, -normal.y);
}

__device__ float dot(const float3& a, const float3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 cross(const float3& a, const float3& b) {
    return make_float3(
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    );
}

__device__ float length(const float3& v) {
    return sqrtf(dot(v, v));
}

__device__ float3 reflect(const float3& v, const float3& n) {
    return (2 * dot(n, v)) * n - v;
}

__device__ float3 normalize(const float3& v) {
    return v * (1.0f / length(v));
}

__device__ float triangle_area(const float3& v0, const float3& v1, const float3& v2) {
    auto n = cross(v1 - v0, v2 - v0);
    return length(n) * 0.5f;
}

__device__ float3 sample_triangle(float u, float v, const float3& v0, const float3& v1, const float3& v2) {
    if (u + v >= 1.0f) {
        u = 1.0f - u;
        v = 1.0f - v;
    }
    return v0 * (1.0f - u - v) + v1 * u + v2 * v;
}

__device__ float luminance(const float3& color) {
    return color.x * 0.2126f + color.y * 0.7152f + color.z * 0.0722f;
}

__device__ float russian_roulette(const float3& color, float clamp) {
    auto prob = 2.0f * luminance(color);
    return prob > clamp ? clamp : prob;
}

template <typename T> __device__ T min(T a, T b) { return a < b ? a : b; }
template <typename T> __device__ T max(T a, T b) { return a > b ? a : b; }
template <typename T> __device__ T clamp(T a, T b, T c) { return min(c, max(a, b)); }
template <typename T, typename U> __device__ T lerp(T a, T b, U k) { return (1.0f - k) * a + k * b; }

// Textures ------------------------------------------------------------------------

__device__ float3 rgba32_to_float3(uint pix) {
    auto r =  pix        & 0xFF;
    auto g = (pix >>  8) & 0xFF;
    auto b = (pix >> 16) & 0xFF;
    auto inv = 1.0f / 255.0f;
    return make_float3(r * inv, g * inv, b * inv);
}

__device__ float3 lookup_texture(int id, float2 uv) {
    auto pix = optix::rtTex2D<float4>(id, uv.x, uv.y);
    return make_float3(pix.x, pix.y, pix.z);
}

// Mesh ----------------------------------------------------------------------------

RT_PROGRAM void mesh_intersect(int prim_id)
{
    auto ids = index_buffer[prim_id];
    auto v0 = vertex_buffer[ids.x];
    auto v1 = vertex_buffer[ids.y];
    auto v2 = vertex_buffer[ids.z];
    float3 fn;
    float t, u, v;
    if (optix::intersect_triangle(ray, v0, v1, v2, fn, t, u, v) && rtPotentialIntersection(t)) {
        attr_texcoord    = (1.0f - u - v) * texcoord_buffer[ids.x] + u * texcoord_buffer[ids.y] + v * texcoord_buffer[ids.z];
        attr_normal      = (1.0f - u - v) * normal_buffer  [ids.x] + u * normal_buffer  [ids.y] + v * normal_buffer  [ids.z];
        attr_face_normal = fn;
        attr_index       = make_int3(ids.x, ids.y, ids.z);
        attr_material    = ids.w;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void mesh_bounds(int prim_id, float result[6]) {
    auto ids = index_buffer[prim_id];
    auto v0 = vertex_buffer[ids.x];
    auto v1 = vertex_buffer[ids.y];
    auto v2 = vertex_buffer[ids.z];
    result[0] = fminf(v0.x, fminf(v1.x, v2.x));
    result[1] = fminf(v0.y, fminf(v1.y, v2.y));
    result[2] = fminf(v0.z, fminf(v1.z, v2.z));
    result[3] = fmaxf(v0.x, fmaxf(v1.x, v2.x));
    result[4] = fmaxf(v0.y, fmaxf(v1.y, v2.y));
    result[5] = fmaxf(v0.z, fmaxf(v1.z, v2.z));
}

// Materials -----------------------------------------------------------------------

__device__ float phong_interp(const float3& kd, const float3& ks) {
    float lum_ks = luminance(ks);
    float lum_kd = luminance(kd);
    return lum_ks + lum_kd == 0 ? 0.0f : lum_ks / (lum_ks + lum_kd);
}

__device__ float specular_sample_pdf(float ns, const float3& normal, const float3& out_dir, const float3& in_dir) {
    auto cos = fmaxf(dot(in_dir, reflect(out_dir, normal)), 0.0f);
    return powf(cos, ns) * (ns + 1.0f) * (1.0f / (2.0f * PI));
}

__device__ float diffuse_sample_pdf(const float3& normal, const float3& in_dir) {
    return fmaxf(dot(in_dir, normal), 0.0f) * (1.0f / PI);
}

__device__ float phong_sample_pdf(const Material& mat, const float3& normal, const float2& uv, const float3& out_dir, const float3& in_dir) {
    auto ks = mat.map_ks >= 0 ? lookup_texture(mat.map_ks, uv) : mat.ks;
    auto kd = mat.map_kd >= 0 ? lookup_texture(mat.map_kd, uv) : mat.kd;
    return lerp(diffuse_sample_pdf(normal, in_dir), specular_sample_pdf(mat.ns, normal, out_dir, in_dir), phong_interp(kd, ks));
}

__device__ float3 eval_specular_bsdf(const float3& ks, float ns, const float3& normal, const float3& out_dir, const float3& in_dir) {
    auto cos = fmaxf(dot(in_dir, reflect(out_dir, normal)), 0.0f);
    return ks * powf(cos, ns) * (ns + 2.0f) * (1.0f / (2.0f * PI));
}

__device__ float3 eval_diffuse_bsdf(const float3& kd) {
    return kd * (1.0f / PI);
}

__device__ float3 eval_phong_bsdf(const Material& mat, const float3& normal, const float2& uv, const float3& out_dir, const float3& in_dir) {
    auto ks = mat.map_ks >= 0 ? lookup_texture(mat.map_ks, uv) : mat.ks;
    auto kd = mat.map_kd >= 0 ? lookup_texture(mat.map_kd, uv) : mat.kd;
    return lerp(eval_diffuse_bsdf(kd), eval_specular_bsdf(ks, mat.ns, normal, out_dir, in_dir), phong_interp(kd, ks));
}

__device__ MaterialSample sample_diffuse_bsdf(const float3& kd, const float3& normal, uint& rnd, const float3& out_dir) {
    // Cosine hemisphere sampling
    auto u = randf(rnd);
    auto v = randf(rnd);
    auto cos = sqrtf(1.0f - v);
    auto sin = sqrtf(v);
    auto phi = 2.0f * PI * u;

    float3 tangent, bitangent;
    gen_local_coords(normal, tangent, bitangent);

    MaterialSample sample;
    sample.dir = sin * cosf(phi) * tangent + sin * sinf(phi) * bitangent + cos * normal;

    sample.color = kd * (1.0f / PI);
    sample.pdf   = cos * (1.0f / PI);
    sample.cos   = cos;
    return sample;
}

__device__ MaterialSample sample_specular_bsdf(const float3& ks, float ns, const float3& normal, uint& rnd, const float3& out_dir) {
    // Cosine-power hemisphere sampling
    auto u = randf(rnd);
    auto v = randf(rnd);
    auto reflect_out = reflect(out_dir, normal);
    auto cos = powf(v, 1.0f / (ns + 1.0f));
    auto sin = sqrtf(1.0f - cos * cos);
    auto phi = 2.0f * PI * u;

    float3 tangent, bitangent;
    gen_local_coords(reflect_out, tangent, bitangent);

    MaterialSample sample;
    sample.dir = sin * cosf(phi) * tangent + sin * sinf(phi) * bitangent + cos * reflect_out;

    auto lobe = powf(cos, ns) * (1.0f / (2.0f * PI));

    sample.color = ks * lobe * (ns + 2.0f);
    sample.pdf   = lobe * (ns + 1.0f);
    sample.cos   = fmaxf(dot(sample.dir, normal), 0.0f);
    return sample;
}

__device__ MaterialSample sample_phong_bsdf(const Material& mat, const float3& normal, const float3& face_normal, const float2& uv, uint& rnd, const float3& out_dir) {
    auto ks = mat.map_ks >= 0 ? lookup_texture(mat.map_ks, uv) : mat.ks;
    auto kd = mat.map_kd >= 0 ? lookup_texture(mat.map_kd, uv) : mat.kd;
    float ns = mat.ns;
    float k = phong_interp(kd, ks);
    bool use_kd = randf(rnd) >= k;
    MaterialSample sample;

    sample = use_kd ? sample_diffuse_bsdf(kd, normal, rnd, out_dir) : sample_specular_bsdf(ks, ns, normal, rnd, out_dir);

    if (sample.pdf <= 0.0f || dot(sample.dir, face_normal) <= 0.0f) {
        sample.pdf = 1.0f;
        sample.color = make_float3(0.0f, 0.0f, 0.0f);
    } else if (use_kd) {
        sample.color = lerp(sample.color, eval_specular_bsdf(ks, ns, normal, out_dir, sample.dir), k);
        sample.pdf   = lerp(sample.pdf,   specular_sample_pdf(ns, normal, out_dir, sample.dir),    k);
    } else {
        sample.color = lerp(eval_diffuse_bsdf(kd),                  sample.color, k);
        sample.pdf   = lerp(diffuse_sample_pdf(normal, sample.dir), sample.pdf,   k);
    }
    return sample;
}

__device__ MaterialSample sample_mirror_bsdf(const Material& mat, const float3& normal, const float3& face_normal, const float3& out_dir) {
    MaterialSample sample;
    sample.cos   = 1.0f;
    sample.pdf   = 1.0f;
    sample.dir   = reflect(out_dir, normal);
    sample.color = dot(sample.dir, face_normal) <= 0 ? make_float3(0.0f, 0.0f, 0.0f) : mat.ks;
    return sample;
}

__device__ float fresnel_factor(float k, float cos_i, float cos_t) {
    const float R_s = (k * cos_i - cos_t) / (k * cos_i + cos_t);
    const float R_p = (cos_i - k * cos_t) / (cos_i + k * cos_t);
    return (R_s * R_s + R_p * R_p) * 0.5f;
}

__device__ MaterialSample sample_glass_bsdf(bool entering, const Material& mat, const float3& normal, const float3& face_normal, uint& rnd, const float3& out_dir) {
    auto n1 = 1.0f;
    auto n2 = mat.ni;
    if (!entering) {
        auto tmp = n1;
        n1 = n2;
        n2 = tmp;
    }
    auto n = n1 / n2;

    auto cos_incoming = dot(out_dir, normal);
    auto cos2_transmitted = 1.0f - n * n * (1.0f - cos_incoming * cos_incoming);

    if (cos2_transmitted > 0.0f) {
        // Refraction
        auto cos_transmitted = sqrtf(cos2_transmitted);
        auto F = fresnel_factor(n, cos_incoming, cos_transmitted);
        if (randf(rnd) > F) {
            auto t = normal * (n * cos_incoming - cos_transmitted) - out_dir * n;
            auto color = dot(t, face_normal) >= 0 ? make_float3(0.0f, 0.0f, 0.0f) : mat.tf;
            MaterialSample sample;
            sample.cos   = 1.0f;
            sample.pdf   = 1.0f;
            sample.dir   = t;
            sample.color = color;
            return sample;
        }
    }

    // Reflection
    return sample_mirror_bsdf(mat, normal, face_normal, out_dir);
}

// OptiX programs ------------------------------------------------------------------

RT_PROGRAM void path_trace() {
    auto x = launch_index.x;
    auto y = launch_index.y;
    auto samples = samples_per_pixel;
    auto result  = make_float3(0.0f, 0.0f, 0.0f);

    do { 
        // Generate ray
        uint rnd = fnv_hash(fnv_init(), x);
        rnd = fnv_hash(rnd, y);
        rnd = fnv_hash(rnd, samples);
        rnd = fnv_hash(rnd, frame_number);
        auto kx = (x + randf(rnd)) * (2.0f / film_width)  - 1.0f;
        auto ky = 1.0f - (y + randf(rnd)) * (2.0f / film_height);
        auto ray_org = cam_eye;
        auto ray_dir = normalize(cam_dir + cam_right * (cam_dim.x * kx) + cam_up * (cam_dim.y * ky));

        RayState ray_state;
        ray_state.depth   = 0;
        ray_state.rnd     = rnd;
        ray_state.contrib = make_float3(1.0f, 1.0f, 1.0f);
        ray_state.color   = make_float3(0.0f, 0.0f, 0.0f);
        ray_state.mis     = 0.0f;
        ray_state.done    = false;

        while (true) {
            optix::Ray ray;
            ray.direction = ray_dir;
            ray.origin    = ray_org;
            ray.tmin      = offset;
            ray.tmax      = FLT_MAX;
            ray.ray_type  = 0;
            rtTrace(top_object, ray, ray_state);

            if (ray_state.done)
                break;

            ray_org = ray_state.next_org;
            ray_dir = ray_state.next_dir;
        }

        result += ray_state.color;
    } while (--samples);

    frame_buffer[y * film_width + x] += result * (1.0f / samples_per_pixel);
}

rtDeclareVariable(RayState, ray_state, rtPayload, );
RT_PROGRAM void closest_hit() {
    auto normal      = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));
    auto face_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_face_normal));
    auto rnd         = ray_state.rnd;
    auto color       = ray_state.color;

    // flip normal as necessary
    bool entering = dot(ray.direction, face_normal) <= 0.0f;
    if (!entering)
        face_normal = -face_normal;
    if (dot(ray.direction, normal) > 0.0f)
        normal = -normal;

    auto& mat = materials[attr_material];
    auto out_dir = -ray.direction;

    // Handle emissive materials
    auto ke = mat.ke;
    if (entering && ke.x != 0 && ke.y != 0 && ke.z != 0) {
        auto index    = attr_index;
        auto v0       = vertex_buffer[index.x];
        auto v1       = vertex_buffer[index.y];
        auto v2       = vertex_buffer[index.z];
        auto pdf_area = 1.0f / triangle_area(v0, v1, v2);
        auto next_mis = ray_state.mis * ray_dist * ray_dist / dot(out_dir, normal);
        auto weight   = 1.0f / (1.0f + next_mis * pdf_lightpick * pdf_area);
        color += ke * ray_state.contrib * weight;
    }

    auto uv = attr_texcoord;
    auto intr_point = ray.origin + ray.direction * ray_dist;

    // Direct illumination is not possible for glass and mirrors
    if (mat.illum != 5 && mat.illum != 7) {
        // Sample a point on a light
        auto& light      = lights[(randi(rnd) & 0x7FFFFFFF) % num_lights];
        auto light_point = sample_triangle(randf(rnd), randf(rnd), light.v0, light.v1, light.v2);
        auto light_dir   = light_point - intr_point;
        auto visibility  = dot(light_dir, normal);
        auto cos_light   = -dot(light.normal, light_dir);

        if (visibility > 0 && cos_light > 0) {
            auto inv_light_dist  = 1.0f / length(light_dir);
            auto inv_light_dist2 = inv_light_dist * inv_light_dist;

            auto in_dir = light_dir * inv_light_dist;

            auto pdf_material  = phong_sample_pdf(mat, normal, uv, out_dir, in_dir);
            auto pdf_light     = light.inv_area * pdf_lightpick;
            auto inv_pdf_light = 1.0f / pdf_light;

            cos_light *= inv_light_dist;
            auto cos_surface = visibility * inv_light_dist;

            auto weight = 1.0f / (1.0f + pdf_material * cos_light * inv_light_dist2 * inv_pdf_light);
            auto geom_factor = cos_surface * cos_light * inv_light_dist2 * inv_pdf_light;

            auto contrib = light.intensity * ray_state.contrib * eval_phong_bsdf(mat, normal, uv, out_dir, in_dir);

            ShadowRayState shadow_ray_state;
            shadow_ray_state.in_shadow = false;
            optix::Ray shadow_ray;
            shadow_ray.origin    = intr_point;
            shadow_ray.direction = light_dir;
            shadow_ray.tmin      = offset;
            shadow_ray.tmax      = 1.0f - offset;
            shadow_ray.ray_type  = 1;
            rtTrace(top_object, shadow_ray, shadow_ray_state);

            if (!shadow_ray_state.in_shadow)
                color += contrib * (geom_factor * weight);
        }
    }

    // Write the new color to the state only once here
    ray_state.color = color;

    // Russian Roulette
    auto rr_prob = russian_roulette(ray_state.contrib, 0.75f);
    if (ray_state.depth >= max_path_depth || randf(rnd) >= rr_prob) {
        ray_state.done = true;
        return;
    }

    MaterialSample sample;
    bool specular = false;
    switch (mat.illum) {
        case 5: // Mirror
            sample = sample_mirror_bsdf(mat, normal, face_normal, out_dir);
            specular = true;
            break;
        case 7: // Glass
            sample = sample_glass_bsdf(entering, mat, normal, face_normal, rnd, out_dir);
            specular = true;
            break;
        default: // Corrected Phong
            sample = sample_phong_bsdf(mat, normal, face_normal, uv, rnd, out_dir);
            break; 
    }

    // Update ray state
    ray_state.depth    ++;
    ray_state.contrib  *= sample.color * (sample.cos / (sample.pdf * rr_prob));
    ray_state.mis      = specular ? 0.0f : 1.0f / sample.pdf;
    ray_state.next_org = intr_point;
    ray_state.next_dir = sample.dir;
    ray_state.rnd      = rnd;
}

rtDeclareVariable(ShadowRayState, shadow_ray_state, rtPayload, );
RT_PROGRAM void shadow() {
    shadow_ray_state.in_shadow = true;
    rtTerminateRay();
}

RT_PROGRAM void exception() {
    auto bad_color = make_float3(1.0f, 0.0f, 1.0f);
    frame_buffer[launch_index.x + launch_index.y * film_width] = bad_color;
}

RT_PROGRAM void miss() {
    ray_state.done = true;
}
